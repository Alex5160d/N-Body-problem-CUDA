#include "hip/hip_runtime.h"
/*
 * bodySystemAbstract.cu
 *
 *  Created on: Dec 4, 2013
 *      Author: alex
 *
 */

#include "bodySystemAbstract.cuh"
#include <algorithm>

/* ************************************************************************** *
 * BodySystemAbstract : methods
 * ************************************************************************** */

// Give random values to each body
void BodySystemAbstract::fillBodies() {
	/*
	 *	pos and vel are arrays of float4 to coalesce memory access
	 *	but those are local variable so we'll preserve registers space using float3 and float
	 */
	float3 point;
	float mass;
	int i = 0;

	// without this scale our bodies will be to close when their number is high
	float scale = 10 * std::max<float>(1.0f, mNumBodies / (1024.0f));
	while (i < mNumBodies) {
		// We try some random positions in intervals [-1;1]
		point.x = rand() / (float) RAND_MAX * 2 - 1;
		point.y = rand() / (float) RAND_MAX * 2 - 1;
		point.z = rand() / (float) RAND_MAX * 2 - 1;

		/*
		 *	we stay in a radius of 1
		 */
		if (distance(point, point) > 1)	//	Euclidean distance
			continue; //	try again

		// our point is alive \o/
		mass = (rand() / (float) (RAND_MAX / 5) + 1) * 1e-6; // mass (arbitrary value between 1*10-6 and 5*10-6)

		mPos[i].w = mass;
		mPos[i].x = point.x * scale;
		mPos[i].y = point.y * scale;
		mPos[i].z = point.z * scale;

		mVel[i].w = 1 / mass; // inverse mass (to compute acceleration)
		// our bodies have no speed at the beginning
		mVel[i].x = 0;
		mVel[i].y = 0;
		mVel[i].z = 0;

		// 'cause color is everything \o/
		mColor[i].w = rand() / (float) RAND_MAX;	// red
		mColor[i].x = rand() / (float) RAND_MAX;	//	green
		mColor[i].y = rand() / (float) RAND_MAX;	//	blue
		mColor[i].z = 1.0f; //	alpha
		i++;
	}
}
