#include "hip/hip_runtime.h"
/*
 * bodySystemCpu.cu
 *
 *  Created on: Dec 4, 2013
 *      Author: alex
 *
 */

#include "bodySystemCpu.cuh"

/* ************************************************************************** *
 * bodySystemCpu : Public methods
 * ************************************************************************** */

void BodySystemCPU::update() {
	integrateSys();
}

float4 *BodySystemCPU::getArray(BodyArray array) {
	switch (array) {
	default:
	case BODYSYSTEM_POSITION:
		return mPos;
	case BODYSYSTEM_VELOCITY:
		return mVel;
	case BODYSYSTEM_COLOR:
			return mColor;
	}
}

/* ************************************************************************** *
 * bodySystemCpu : Protected methods
 * ************************************************************************** */

void BodySystemCPU::_initialize() {
	mPos = new float4[mNumBodies];
	mVel = new float4[mNumBodies];
	mColor = new float4[mNumBodies];
	mAcc = new float3[mNumBodies];

	memset(mPos, 0, mNumBodies * sizeof(float4));
	memset(mVel, 0, mNumBodies * sizeof(float4));
	memset(mColor, 0, mNumBodies * sizeof(float4));
	memset(mAcc, 0, mNumBodies * sizeof(float3));
}

void BodySystemCPU::_finalize() {
	delete[] mPos;
	delete[] mVel;
	delete[] mColor;
	delete[] mAcc;
}

/* ************************************************************************** *
 * bodySystemCpu : Private methods
 * ************************************************************************** */

void BodySystemCPU::computeGrav() {
	// loop on every body \o/
	for (int i = 0; i < mNumBodies; i++) {
		float3 acc = { 0, 0, 0 };
		//	for each body, we compute his interaction with each other
		for (int j = 0; j < mNumBodies; j++)
			bodyInterac(acc, mPos[i], mPos[j]);

		//	the new acceleration
		mAcc[i] = acc;
	}
}

void BodySystemCPU::integrateSys() {
	computeGrav();
	/*
	 * we need those local variables make the computation easier
	 * by dividing between mass and position/velocity
	 */
	float3 lpos, lvel;
	for (int i = 0; i < mNumBodies; ++i) {
		// we save the old values
		lpos.x = mPos[i].x;
		lpos.y = mPos[i].y;
		lpos.z = mPos[i].z;

		lvel.x = mVel[i].x;
		lvel.y = mVel[i].y;
		lvel.z = mVel[i].z;

		// new velocity = old velocity + acceleration * deltaTime
		lvel = lvel + scalevec(mAcc[i], DELTA_TIME);

		// new position = old position + velocity * deltaTime
		lpos = lpos + scalevec(lvel, DELTA_TIME);

		mPos[i].x = lpos.x;
		mPos[i].y = lpos.y;
		mPos[i].z = lpos.z;

		mVel[i].x = lvel.x;
		mVel[i].y = lvel.y;
		mVel[i].z = lvel.z;
	}
}

void BodySystemCPU::bodyInterac(float3& accel, float4 const& posFirst, float4 const& posSec) {
	float3 r;

	// the vector going from body 1 to 0
	r.x = posSec.x - posFirst.x;
	r.y = posSec.y - posFirst.y;
	r.z = posSec.z - posFirst.z;

	//	see gravity law
	accel = accel + scalevec(scalevec(r, (float) posSec.w * (float) pow(rsqrt(dot(r, r) + SOFTENINGSQUARED), 3)), 9.81f);
}
