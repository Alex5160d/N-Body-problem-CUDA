#include "hip/hip_runtime.h"
/*
 * bodySystemCuda.cu
 *
 *  Created on: Dec 8, 2013
 *      Author: alex
 *
 */

#include "bodySystemCuda.cuh"

#include <cuda_gl_interop.h>
#include <algorithm>

/* ************************************************************************** *
 * bodySystemCUDA : Used CUDA functions
 * ************************************************************************** */

//	Compute the interaction between two bodies
__device__
float3 bodyInterac(float3 accel, float4 posFirst, float4 posSec) {
	float3 r;

	// the vector from body 1 to 0
	r.x = posSec.x - posFirst.x;
	r.y = posSec.y - posFirst.y;
	r.z = posSec.z - posFirst.z;

	//	see gravity law
	return (accel
			+ scalevec(scalevec(r,
					(float) posSec.w
							* (float) pow(rsqrt(dot(r, r) + SOFTENINGSQUARED),
									3)), 9.81f));
}

// // compute the new acceleration of all bodies
__device__ float3 computeGrav(float4 bodyPos, float4 *positions,
		int numBodies) {
	extern __shared__ float4 sharedPos[];

	float3 acc = { 0.0f, 0.0f, 0.0f };
	/*
	 * Compute the interaction of our body with all bodies of a block
	 * then do the same with the next block
	 */
	for (int i = 0; i < gridDim.x; i++) {
		// We first need to copy the positions of all bodies in the block
		sharedPos[threadIdx.x] = positions[i * blockDim.x + threadIdx.x];
		// wait for the others to have copied a body
		__syncthreads();

		// Then we'll start computing the interaction with the block
		for (unsigned int counter = 0; counter < blockDim.x; counter++)
			acc = bodyInterac(acc, bodyPos, sharedPos[counter]);
		//	When we use the shared memory we always have to sync at the end
		__syncthreads();
	}

	return acc;
}

// use the method above and compute the new positions and velocities
__global__ void integrateSys(float4* newPos, float4* oldPos, float4* vel,
		int numBodies) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	float4 position = oldPos[index];
	float3 accel = computeGrav(position, oldPos, numBodies);

	// Now that we have the new acceleration we can update all the values
	float4 velocity = vel[index];

	// velocity is an integrator of acceleration and position is an integrator of velocity
	velocity.x += accel.x * DELTA_TIME;
	velocity.y += accel.y * DELTA_TIME;
	velocity.z += accel.z * DELTA_TIME;

	position.x += velocity.x * DELTA_TIME;
	position.y += velocity.y * DELTA_TIME;
	position.z += velocity.z * DELTA_TIME;

	/*
	 * store new position and velocity in a new
	 * location for the position because of the interaction with opengl
	 */
	newPos[index] = position;
	vel[index] = velocity;
}

/* ************************************************************************** *
 * bodySystemCUDA : Public methods
 * ************************************************************************** */

BodySystemCUDA::BodySystemCUDA(int numBodies) :
		BodySystemAbstract(numBodies), mCurrentRead(0), mCurrentWrite(1), mDVel(
				0) {
	/*
	 * Less blocks means less copies from global to shared memory
	 *
	 * 1 thread = 1 position and 1 position = 1 float4 (4*4 bytes)
	 * with 16KB of shared memory, max = 1024 bodies
	 */
	if(numBodies <= 1024)	// We have enough shared memory for all bodies (numBodies*sizeof(float4)
	{
		mBlockSize = numBodies;
	}
	else	// We divide our problem in blocks of same size and taking in account the size of a wrap
	{
		for(mBlockSize=1024; mBlockSize>=32; mBlockSize--)
			if(mBlockSize%32==0 && numBodies%mBlockSize==0)
				break;
	}
	// since the number of bodies is a multiple of 32, we'll have no remainder
	mNumBlocks = mNumBodies / mBlockSize;
	sharedMemSize = mBlockSize * sizeof(float4);
	_initialize();
}

void BodySystemCUDA::update() {

	integrateNBodySystem();

	std::swap(mCurrentRead, mCurrentWrite);
}

void BodySystemCUDA::setArrays() {
	//	we bind and fill the position buffer for cuda and opengl
	glBindBuffer(GL_ARRAY_BUFFER, mPbo[mCurrentRead]);
	glBufferSubData(GL_ARRAY_BUFFER, 0, sizeof(float4) * mNumBodies, mPos);
	//	done
	glBindBuffer(GL_ARRAY_BUFFER, 0);

	/*
	 * the velocity won't need to be transfered to opengl
	 * so a cudamemcpy is enough
	 */
	hipMemcpy(mDVel, mVel, mNumBodies * sizeof(float4),
			hipMemcpyHostToDevice);
}

/* ************************************************************************** *
 * bodySystemCUDA : Protected methods
 * ************************************************************************** */

void BodySystemCUDA::_initialize() {
	unsigned int memSize = sizeof(float4) * mNumBodies;

	mPos = new float4[mNumBodies];
	mVel = new float4[mNumBodies];
	mColor = new float4[mNumBodies];

	memset(mPos, 0, mNumBodies * sizeof(float4));
	memset(mVel, 0, mNumBodies * sizeof(float4));
	memset(mColor, 0, mNumBodies * sizeof(float4));

	// create the position pixel buffer objects for rendering
	glGenBuffers(2, (GLuint *) mPbo);

	/*
	 * we fill the buffer and create the link with cuda
	 * at first old and new positions are the same so we can compute
	 * on a buffer while opengl is rendering the other
	 */
	for (int i = 0; i < 2; ++i) {
		glBindBuffer(GL_ARRAY_BUFFER, mPbo[i]);
		glBufferData(GL_ARRAY_BUFFER, memSize, mPos, GL_DYNAMIC_DRAW);
		glBindBuffer(GL_ARRAY_BUFFER, 0);
		hipGraphicsGLRegisterBuffer(&mCGRes[i], mPbo[i],
				cudaGraphicsMapFlagsNone);
	}
	hipMalloc((void **) &mDVel, memSize);
}

void BodySystemCUDA::_finalize() {

	delete[] mPos;
	delete[] mVel;
	delete[] mColor;

	hipFree(mDVel);

	hipGraphicsUnregisterResource(mCGRes[0]);
	hipGraphicsUnregisterResource(mCGRes[1]);
	glDeleteBuffers(2, (const GLuint *) mPbo);
}

/* ************************************************************************** *
 * bodySystemCUDA : Private methods
 * ************************************************************************** */

void BodySystemCUDA::integrateNBodySystem() {
	/*
	 * Between each update the write and read buffers are swapped
	 * so we need to remap
	 */
	cudaGraphicsResourceSetMapFlags(mCGRes[mCurrentRead],
			cudaGraphicsMapFlagsReadOnly);
	cudaGraphicsResourceSetMapFlags(mCGRes[1 - mCurrentRead],
			cudaGraphicsMapFlagsWriteDiscard);
	hipGraphicsMapResources(2, mCGRes, 0);
	size_t bytes;
	hipGraphicsResourceGetMappedPointer((void **) &mDeviceCpPos[mCurrentRead],
			&bytes, mCGRes[mCurrentRead]);
	hipGraphicsResourceGetMappedPointer(
			(void **) &(mDeviceCpPos[1 - mCurrentRead]), &bytes,
			mCGRes[1 - mCurrentRead]);

	// launch the actual computation
	integrateSys<<<mNumBlocks, mBlockSize, sharedMemSize>>>(
			mDeviceCpPos[1 - mCurrentRead], mDeviceCpPos[mCurrentRead], mDVel,
			mNumBodies);
	hipGraphicsUnmapResources(2, mCGRes, 0);
}
