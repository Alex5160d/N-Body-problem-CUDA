#include "hip/hip_runtime.h"
/*
 * renderer.cu
 *
 *  Created on: Dec 8, 2013
 *      Author: alex
 *
 */

#include "renderer.cuh"

/* ************************************************************************** *
 * Renderer : Public methods
 * ************************************************************************** */

// Those are values from research and experiments, should be setted in a more intelligent way...
Renderer::Renderer() :
		mPos(0), mNumParticles(0), mSpriteSize(2.0f), mVertexShader(0), mVertexShaderPoints(
				0), mPixelShader(0), mProgramSprites(0), mTexture(0), mPbo(0), mVboColor(
				0) {
	mBaseColor[0] = 1.0f;
	mBaseColor[1] = 0.6f;
	mBaseColor[2] = 0.3f;
	mBaseColor[3] = 1.0f;
	_initGL();
}

Renderer::~Renderer() {
	mPos = 0;
}

// bind and fill the opengl buffer using our bodies positions
void Renderer::setPositions(float4 *pos, int numParticles) {
	mPos = pos;
	mNumParticles = numParticles;

	if (!mPbo) {
		glGenBuffers(1, (GLuint *) &mPbo);
	}

	glBindBuffer(GL_ARRAY_BUFFER_ARB, mPbo);
	glBufferData(GL_ARRAY_BUFFER_ARB, numParticles * sizeof(float4), pos,
			GL_STATIC_DRAW_ARB);
	glBindBuffer(GL_ARRAY_BUFFER_ARB, 0);
}

//	same for the colors
void Renderer::setColors(float4 *color, int numParticles) {
	glBindBuffer(GL_ARRAY_BUFFER_ARB, mVboColor);
	glBufferData(GL_ARRAY_BUFFER_ARB, numParticles * sizeof(float4), color,
			GL_STATIC_DRAW_ARB);
	glBindBuffer(GL_ARRAY_BUFFER_ARB, 0);
}

void Renderer::setPBO(unsigned int pbo, int numParticles) {
	mPbo = pbo;
	mNumParticles = numParticles;
}

void Renderer::display() {
	// setup point sprites
	glEnable(GL_POINT_SPRITE_ARB);
	glTexEnvi(GL_POINT_SPRITE_ARB, GL_COORD_REPLACE_ARB, GL_TRUE);
	glEnable(GL_VERTEX_PROGRAM_POINT_SIZE_NV);
	glPointSize(mSpriteSize);
	glBlendFunc(GL_SRC_ALPHA, GL_ONE);
	glEnable(GL_BLEND);
	glDepthMask(GL_FALSE);

	glUseProgram(mProgramSprites);
	GLuint texLoc = glGetUniformLocation(mProgramSprites, "splatTexture");
	glUniform1i(texLoc, 0);

	glActiveTextureARB(GL_TEXTURE0_ARB);
	glBindTexture(GL_TEXTURE_2D, mTexture);

	glColor3f(1, 1, 1);
	glSecondaryColor3fv(mBaseColor);

	_drawPoints();

	glUseProgram(0);

	glDisable(GL_POINT_SPRITE_ARB);
	glDisable(GL_BLEND);
	glDepthMask(GL_TRUE);
}

/* ************************************************************************** *
 * Renderer : Protected methods
 * ************************************************************************** */

void Renderer::_drawPoints() {
	if (!mPbo) {
		glBegin(GL_POINTS);
		{
			//	A small trick to draw our vertex3 with float4
			int k = 0;
			for (int i = 0; i < mNumParticles; ++i) {
				glVertex3fv((float*) &mPos[k]);
				k += 4;
			}
		}
		glEnd();
	} else {	// so much simple with pbo
		glEnableClientState(GL_VERTEX_ARRAY);

		glBindBufferARB(GL_ARRAY_BUFFER_ARB, mPbo);

		glVertexPointer(4, GL_FLOAT, 0, 0);

		glEnableClientState(GL_COLOR_ARRAY);
		glBindBufferARB(GL_ARRAY_BUFFER_ARB, mVboColor);
		glColorPointer(4, GL_FLOAT, 0, 0);

		glDrawArrays(GL_POINTS, 0, mNumParticles);
		glBindBufferARB(GL_ARRAY_BUFFER_ARB, 0);
		glDisableClientState(GL_VERTEX_ARRAY);
		glDisableClientState(GL_COLOR_ARRAY);
	}
}

const char vertexShaderPoints[] =
		{
				"void main()                                                            						\n"
						"{                                                                      				\n"
						"    vec4 vertex = vec4(gl_Vertex.xyz, 1.0);  			                				\n"
						"    gl_Position = gl_ProjectionMatrix * gl_ModelViewMatrix * vertex;   				\n"
						"    gl_FrontColor = gl_Color;                                          				\n"
						"}                                                                      				\n" };

const char vertexShader[] =
		{
				"void main()                                                            						\n"
						"{                                                                      				\n"
						"    float pointSize = 500.0 * gl_Point.size;                           				\n"
						"    vec4 vertex = gl_Vertex;															\n"
						"    vertex.w = 1.0;																	\n"
						"    vec3 pos_eye = vec3 (gl_ModelViewMatrix * vertex);                 				\n"
						"    gl_PointSize = max(1.0, pointSize / (1.0 - pos_eye.z));            				\n"
						"    gl_TexCoord[0] = gl_MultiTexCoord0;                                				\n"
						"    gl_Position = gl_ProjectionMatrix * gl_ModelViewMatrix * vertex;   				\n"
						"    gl_FrontColor = gl_Color;                                          				\n"
						"    gl_FrontSecondaryColor = gl_SecondaryColor;                        				\n"
						"}                                                                      				\n" };

const char pixelShader[] =
		{
				"uniform sampler2D splatTexture;                                        						\n"
						"void main()                                                            				\n"
						"{                                                                      				\n"
						"    vec4 colorSec = gl_SecondaryColor;                                 				\n"
						"    vec4 color = (0.6 + 0.4 * gl_Color) * texture2D(splatTexture, gl_TexCoord[0].st); 	\n"
						"    gl_FragColor = color * colorSec;													\n"
						"}                                                                      				\n" };

// Will create and attach all the shader to apply on our image
void Renderer::_initGL() {
	mVertexShader = glCreateShader(GL_VERTEX_SHADER);
	mVertexShaderPoints = glCreateShader(GL_VERTEX_SHADER);
	mPixelShader = glCreateShader(GL_FRAGMENT_SHADER);

	const char *vertex = vertexShader;
	const char *pixel = pixelShader;
	glShaderSource(mVertexShader, 1, &vertex, 0);
	glShaderSource(mPixelShader, 1, &pixel, 0);
	const char *vp = vertexShaderPoints;
	glShaderSource(mVertexShaderPoints, 1, &vp, 0);

	glCompileShader(mVertexShader);
	glCompileShader(mVertexShaderPoints);
	glCompileShader(mPixelShader);

	mProgramSprites = glCreateProgram();
	glAttachShader(mProgramSprites, mVertexShader);
	glAttachShader(mProgramSprites, mPixelShader);
	glLinkProgram(mProgramSprites);

	_createTexture();

	glGenBuffers(1, (GLuint *) &mVboColor);
	glBindBuffer(GL_ARRAY_BUFFER_ARB, mVboColor);
	glBufferData(GL_ARRAY_BUFFER_ARB, mNumParticles * sizeof(float4), 0,
			GL_STATIC_DRAW_ARB);
	glBindBuffer(GL_ARRAY_BUFFER_ARB, 0);
}

/*
 *	Build a beautiful texture to have a gaussian blur on our bodies
 *
 */

// Gaussian approximation
inline float evalHermite(float dist) {
	return (2 * pow(dist, 3) - 3 * pow(dist, 2) + 1);
}

// Create the gaussian blur applied to each body
uchar4 *createGaussianMap(int resolution) {
	/*
	 * Red/Green/Blue/Alpha values at each coordinate (at a distance of 1 from the body)
	 * with a step of 2/resolution
	 */
	uchar4 *texturMatrix = new uchar4[resolution * resolution];
	// the x coordinate, y coordinate, y squarred to compute the distance
	float x, y, ySquarred, dist;
	// The y and x value are between -1 and 1 (coordinates system centered on the body)
	float step = 2.0f / resolution;
	y = -1.0f;
	//
	int j = 0;

	for (int a = 0; a < resolution; a++, y += step) {
		ySquarred = pow(y, 2);
		x = -1.0f;

		for (int b = 0; b < resolution; b++, x += step, j++) {
			dist = (float) sqrtf(pow(x, 2) + ySquarred); // euclidean distance

			if (dist > 1) // we keep a radius of 1
				dist = 1;
			// we compute the color values for this distance
			texturMatrix[j].w = texturMatrix[j].x = texturMatrix[j].y =
					texturMatrix[j].z =
							(unsigned char) (evalHermite(dist) * 255);
		}
	}

	return (texturMatrix);
}

void Renderer::_createTexture(int resolution) {
	uchar4 *colorMap = createGaussianMap(resolution);
	glGenTextures(1, (GLuint *) &mTexture);
	glBindTexture(GL_TEXTURE_2D, mTexture);
	glTexParameteri(GL_TEXTURE_2D, GL_GENERATE_MIPMAP_SGIS, GL_TRUE);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER,
			GL_LINEAR_MIPMAP_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, resolution, resolution, 0, GL_RGBA,
			GL_UNSIGNED_BYTE, colorMap);

}
